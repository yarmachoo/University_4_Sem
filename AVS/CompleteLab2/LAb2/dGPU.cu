
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <chrono>
#include <fstream>
#include <string>

double Y(double x)
{
    return(-std::log(std::abs(2.0*std::sin(x/2.0))));
}

__device__ void atomicAddAndCheck(double* sum, double value, double yVal, double eps, int* metCondition, int idx)
{
    unsigned long long int* addressAsUll = (unsigned long long int*)sum;
    unsigned long long int old = *addressAsUll, assumed;

    do {
        assumed = old;
        double assumedAsDouble = __longlong_as_double(assumed);
        double newValue = assumedAsDouble + value;
        old = atomicCAS(addressAsUll, assumed, __double_as_longlong(newValue));
        if (abs(newValue - yVal) <= eps)
        {
            atomicExch(&metCondition[idx], 1);
        }
    } while (assumed != old);
}

__global__ void computeTerms(double x, double* output, double* sum, double yVal, double eps, int* metCondition,int cycles)
{
    int idx = (threadIdx.x + blockIdx.x * blockDim.x);
    output[idx] = cos(((idx+1)+512*(cycles)) * x)/((idx+1)+512*(cycles));
    atomicAddAndCheck(sum, output[idx], yVal, eps, metCondition, idx);
}

double S(double x, double eps, int& n)
{
    double sum = 0;
    double yVal = Y(x);

    int cycles = 0;

    int* metCondition = new int[512];

    while (true)
    {
        double* dOutput;
        hipMalloc(&dOutput, 512 * sizeof(double));

        double* dSum;
        hipMalloc(&dSum, sizeof(double));
        hipMemcpy(dSum, &sum, sizeof(double), hipMemcpyHostToDevice);

        int* dMetCondition;
        hipMalloc(&dMetCondition, 512 * sizeof(int));

        computeTerms<<<1, 512>>>(x, dOutput, dSum, yVal, eps, dMetCondition, cycles);

        hipMemcpy(&sum, dSum, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(metCondition, dMetCondition, 512 * sizeof(int), hipMemcpyDeviceToHost);

        hipFree(dOutput);
        hipFree(dSum);
        hipFree(dMetCondition);

        for (int i = 0; i < 512; i++)
        {
            if (metCondition[i] == 1)
            {
                n += i;
                break;
            }
        }

        cycles++;

        if (std::abs(sum - yVal) <= eps)
        {
            break;
        }
    }

    delete[] metCondition;

    std::cout << "Cycles: " << cycles << std::endl;

    return sum;
}

int main()
{
    std::string logFileDataName;
    std::string logFileName;

    logFileName = "logCUDA.txt";
    logFileDataName = "logCUDAData.txt";

    double a, b, h, eps;
    // std::cout << "Enter a, b, h, eps: ";
    // std::cin >> a >> b >> h >> eps;
    a = 0.1, b = 0.97, h = 0.001, eps = 0.001;

    std::ofstream logFile(logFileName);
    std::ofstream logFileData(logFileDataName);

    for (double x = a; std::abs(x - b) > eps; x += h)
    {
        int n = 1;

        auto start = std::chrono::high_resolution_clock::now();

        double y = Y(x);
        double s = S(x, eps, n);

        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

        std::cout << "x = " << x << ", Y(x) = " << y << ", S(x) = " << s << ", n = " << n << ", time = " << duration << " ns" << std::endl;
        logFile << "x = " << x << ", Y(x) = " << y << ", S(x) = " << s << ", n = " << n << ", time = " << duration << " ns" << std::endl;
        logFileData << n << " " << duration << std::endl;
    }

    return 0;
}
